
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <math.h> 
using namespace std;

#define PI 3.14159265

// GPU ERROR CHECKING MACRO
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// COMPUTE KERNEL
__global__
void mykern(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] =(a*x[i] + y[i]);
}

int main(void)
{
  cout << "LES CUDA Kernel Test\n" << std::flush;
  int N = 100000000;

  float my_const = 0.5;

  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  gpuErrchk(hipMalloc(&d_x, N*sizeof(float))); 
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 2.0f;
    y[i] = 0.0f;
  }

  cout << "Setup Done\n" << std::flush;

  for( int index = 0; index < 50; index+=1 ) {  
    // COPY MEMORY TO DEVICE
    gpuErrchk( hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice));
    hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

    // Perform KERNEL on all elements
    mykern<<<(N+255)/256, 256>>>(N, my_const, d_x, d_y);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // COPY DEVICE MEMORY TO CPU 
    hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
    if (index%10 ==0){
      cout << " index:" << index << "\n" << std::flush;}
  }

  cout << "GPU Computations Done\n" << std::flush;

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-50.0f));
  printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}

